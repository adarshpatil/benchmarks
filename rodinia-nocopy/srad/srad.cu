#include "hip/hip_runtime.h"
// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <srad.h>

// includes, project
#include <hip/hip_runtime.h>

// includes, kernels
#include <srad_kernel.cu>


#ifdef GEM5_FUSION
#include <stdint.h>
extern "C" {
void m5_work_begin(uint64_t workid, uint64_t threadid);
void m5_work_end(uint64_t workid, uint64_t threadid);
void m5_dump_stats(uint64_t ns_delay, uint64_t ns_period);
}
#endif

void random_matrix(float *I, int rows, int cols);
void runTest( int argc, char** argv);
void usage(int argc, char **argv)
{
	fprintf(stderr, "Usage: %s <rows> <cols> <y1> <y2> <x1> <x2> <lamda> <no. of iter>\n", argv[0]);
	fprintf(stderr, "\t<rows>   - number of rows\n");
	fprintf(stderr, "\t<cols>    - number of cols\n");
	fprintf(stderr, "\t<y1> 	 - y1 value of the speckle\n");
	fprintf(stderr, "\t<y2>      - y2 value of the speckle\n");
	fprintf(stderr, "\t<x1>       - x1 value of the speckle\n");
	fprintf(stderr, "\t<x2>       - x2 value of the speckle\n");
	fprintf(stderr, "\t<lamda>   - lambda (0,1)\n");
	fprintf(stderr, "\t<no. of iter>   - number of iterations\n");

	exit(1);
}
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv)
{

    // BEGIN ADARSH DUMMY LOOP
    int blosum62[24][24] = {
	{ 4, -1, -2, -2,  0, -1, -1,  0, -2, -1, -1, -1, -1, -2, -1,  1,  0, -3, -2,  0, -2, -1,  0, -4},
	{-1,  5,  0, -2, -3,  1,  0, -2,  0, -3, -2,  2, -1, -3, -2, -1, -1, -3, -2, -3, -1,  0, -1, -4},
	{-2,  0,  6,  1, -3,  0,  0,  0,  1, -3, -3,  0, -2, -3, -2,  1,  0, -4, -2, -3,  3,  0, -1, -4},
	{-2, -2,  1,  6, -3,  0,  2, -1, -1, -3, -4, -1, -3, -3, -1,  0, -1, -4, -3, -3,  4,  1, -1, -4},
	{ 0, -3, -3, -3,  9, -3, -4, -3, -3, -1, -1, -3, -1, -2, -3, -1, -1, -2, -2, -1, -3, -3, -2, -4},
	{-1,  1,  0,  0, -3,  5,  2, -2,  0, -3, -2,  1,  0, -3, -1,  0, -1, -2, -1, -2,  0,  3, -1, -4},
	{-1,  0,  0,  2, -4,  2,  5, -2,  0, -3, -3,  1, -2, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
	{ 0, -2,  0, -1, -3, -2, -2,  6, -2, -4, -4, -2, -3, -3, -2,  0, -2, -2, -3, -3, -1, -2, -1, -4},
	{-2,  0,  1, -1, -3,  0,  0, -2,  8, -3, -3, -1, -2, -1, -2, -1, -2, -2,  2, -3,  0,  0, -1, -4},
	{-1, -3, -3, -3, -1, -3, -3, -4, -3,  4,  2, -3,  1,  0, -3, -2, -1, -3, -1,  3, -3, -3, -1, -4},
	{-1, -2, -3, -4, -1, -2, -3, -4, -3,  2,  4, -2,  2,  0, -3, -2, -1, -2, -1,  1, -4, -3, -1, -4},
	{-1,  2,  0, -1, -3,  1,  1, -2, -1, -3, -2,  5, -1, -3, -1,  0, -1, -3, -2, -2,  0,  1, -1, -4},
	{-1, -1, -2, -3, -1,  0, -2, -3, -2,  1,  2, -1,  5,  0, -2, -1, -1, -1, -1,  1, -3, -1, -1, -4},
	{-2, -3, -3, -3, -2, -3, -3, -3, -1,  0,  0, -3,  0,  6, -4, -2, -2,  1,  3, -1, -3, -3, -1, -4},
	{-1, -2, -2, -1, -3, -1, -1, -2, -2, -3, -3, -1, -2, -4,  7, -1, -1, -4, -3, -2, -2, -1, -2, -4},
	{ 1, -1,  1,  0, -1,  0,  0,  0, -1, -2, -2,  0, -1, -2, -1,  4,  1, -3, -2, -2,  0,  0,  0, -4},
	{ 0, -1,  0, -1, -1, -1, -1, -2, -2, -1, -1, -1, -1, -2, -1,  1,  5, -2, -2,  0, -1, -1,  0, -4},
	{-3, -3, -4, -4, -2, -2, -3, -2, -2, -3, -2, -3, -1,  1, -4, -3, -2, 11,  2, -3, -4, -3, -2, -4},
	{-2, -2, -2, -3, -2, -1, -2, -3,  2, -1, -1, -2, -1,  3, -3, -2, -2,  2,  7, -1, -3, -2, -1, -4},
	{ 0, -3, -3, -3, -1, -2, -2, -3, -3,  3,  1, -2,  1, -1, -2, -2,  0, -3, -1,  4, -3, -2, -1, -4},
	{-2, -1,  3,  4, -3,  0,  1, -1,  0, -3, -4,  0, -3, -3, -2,  0, -1, -4, -3, -3,  4,  1, -1, -4},
	{-1,  0,  0,  1, -3,  3,  4, -2,  0, -3, -3,  1, -1, -3, -1,  0, -1, -3, -2, -2,  1,  4, -1, -4},
	{ 0, -1, -1, -1, -2, -1, -1, -1, -1, -1, -1, -1, -1, -1, -2,  0,  0, -2, -1, -1, -1, -1, -1, -4},
	{-4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4, -4,  1}
    };

    int dummyArray[48][48];
    int i,j,k,ctr;
    for ( i=0; i<24; i++)
        for ( j=0; j<24; j++){
            dummyArray[i][j] = blosum62[i][j];
            dummyArray[i+24][j] = blosum62[i][j];
            dummyArray[i][j+24] = blosum62[i][j];
            dummyArray[i+24][j+24] = blosum62[i][j];
         }

    for ( k=1; k<6000; k++) {
        for ( i=1; i<47; i++)
            for ( j=1; j<47; j++)
                dummyArray[i][j] += (dummyArray[i-1][j] + dummyArray[i+1][j]) * (dummyArray[i][j+1] + dummyArray[i][j-1]);

        for ( i=1; i<47; i++)
            for ( j=1; j<47; j++)
                dummyArray[i][j] += (dummyArray[i-1][j] * dummyArray[i+1][j]) + (dummyArray[i][j+1] * dummyArray[i][j-1]);

        for (i=0;i<48; i++) {
            ctr = dummyArray[0][i];
            for ( j=1;j<48; j++) {
                ctr += dummyArray[j][i];
                dummyArray[j][i] = ctr;
            }
        }

		for (i=0;i<48; i++) {
		    ctr = dummyArray[i][0];
		    for ( j=1;j<48; j++) {
		        ctr += dummyArray[i][j];
		        dummyArray[i][j] = ctr;
		    }
        }
    }

    fprintf(stdout, "Begin dummy output\n");
    for ( i=1; i<48; i++)
        fprintf(stdout, "%d ", dummyArray[23][i]);
    fprintf(stdout, "\nEnd of dummy output\n");

    // END ADARSH DUMMY LOOP

    runTest( argc, argv);

    return EXIT_SUCCESS;
}


void
runTest( int argc, char** argv)
{
    int rows, cols, size_I, size_R, niter = 10, iter;
    float *I, *J, lambda, q0sqr, sum, sum2, tmp, meanROI,varROI ;

    float *dN,*dS,*dW,*dE;
#ifdef CPU
	float Jc, G2, L, num, den, qsqr;
	int *iN,*iS,*jE,*jW, k;
	float cN,cS,cW,cE,D;
#endif

#ifdef GPU

	//float *J_cuda;
  //  float *C_cuda;
	//float *E_C, *W_C, *N_C, *S_C;

#endif

	unsigned int r1, r2, c1, c2;
	float *c;



	if (argc == 9)
	{
		rows = atoi(argv[1]);  //number of rows in the domain
		cols = atoi(argv[2]);  //number of cols in the domain
		if ((rows%16!=0) || (cols%16!=0)){
		fprintf(stderr, "rows and cols must be multiples of 16\n");
		exit(1);
		}
		r1   = atoi(argv[3]);  //y1 position of the speckle
		r2   = atoi(argv[4]);  //y2 position of the speckle
		c1   = atoi(argv[5]);  //x1 position of the speckle
		c2   = atoi(argv[6]);  //x2 position of the speckle
		lambda = atof(argv[7]); //Lambda value
		niter = atoi(argv[8]); //number of iterations

	}
    else{
	usage(argc, argv);
    }



	size_I = cols * rows;
    size_R = (r2-r1+1)*(c2-c1+1);

	I = (float *)malloc( size_I * sizeof(float) );
    J = (float *)malloc( size_I * sizeof(float) );
	c  = (float *)malloc(sizeof(float)* size_I) ;



	dN = (float *)malloc(sizeof(float)* size_I) ;
    dS = (float *)malloc(sizeof(float)* size_I) ;
    dW = (float *)malloc(sizeof(float)* size_I) ;
    dE = (float *)malloc(sizeof(float)* size_I) ;

#ifdef CPU

    iN = (int *)malloc(sizeof(unsigned int*) * rows) ;
    iS = (int *)malloc(sizeof(unsigned int*) * rows) ;
    jW = (int *)malloc(sizeof(unsigned int*) * cols) ;
    jE = (int *)malloc(sizeof(unsigned int*) * cols) ;


    for (int i=0; i< rows; i++) {
        iN[i] = i-1;
        iS[i] = i+1;
    }
    for (int j=0; j< cols; j++) {
        jW[j] = j-1;
        jE[j] = j+1;
    }
    iN[0]    = 0;
    iS[rows-1] = rows-1;
    jW[0]    = 0;
    jE[cols-1] = cols-1;

#endif

#ifdef GPU

	//Allocate device memory
    //hipMalloc((void**)& J_cuda, sizeof(float)* size_I);
    //hipMalloc((void**)& C_cuda, sizeof(float)* size_I);
	//hipMalloc((void**)& E_C, sizeof(float)* size_I);
	//hipMalloc((void**)& W_C, sizeof(float)* size_I);
	//hipMalloc((void**)& S_C, sizeof(float)* size_I);
	//hipMalloc((void**)& N_C, sizeof(float)* size_I);


#endif

	printf("Randomizing the input matrix\n");
	//Generate a random matrix
	random_matrix(I, rows, cols);

    for (int k = 0;  k < size_I; k++ ) {
     	J[k] = (float)exp(I[k]) ;
    }
	printf("Start the SRAD main loop\n");
#ifdef GEM5_FUSION
    m5_dump_stats(0, 0);
    m5_work_begin(0, 0);
#endif
 for (iter=0; iter< niter; iter++){
		sum=0; sum2=0;
        for (int i=r1; i<=r2; i++) {
            for (int j=c1; j<=c2; j++) {
                tmp   = J[i * cols + j];
                sum  += tmp ;
                sum2 += tmp*tmp;
            }
        }
        meanROI = sum / size_R;
        varROI  = (sum2 / size_R) - meanROI*meanROI;
        q0sqr   = varROI / (meanROI*meanROI);



#ifdef CPU

		for (int i = 0 ; i < rows ; i++) {
            for (int j = 0; j < cols; j++) {

				k = i * cols + j;
				Jc = J[k];

				// directional derivates
                dN[k] = J[iN[i] * cols + j] - Jc;
                dS[k] = J[iS[i] * cols + j] - Jc;
                dW[k] = J[i * cols + jW[j]] - Jc;
                dE[k] = J[i * cols + jE[j]] - Jc;

                G2 = (dN[k]*dN[k] + dS[k]*dS[k]
                    + dW[k]*dW[k] + dE[k]*dE[k]) / (Jc*Jc);

   		        L = (dN[k] + dS[k] + dW[k] + dE[k]) / Jc;

				num  = (0.5*G2) - ((1.0/16.0)*(L*L)) ;
                den  = 1 + (.25*L);
                qsqr = num/(den*den);

                // diffusion coefficent (equ 33)
                den = (qsqr-q0sqr) / (q0sqr * (1+q0sqr)) ;
                c[k] = 1.0 / (1.0+den) ;

                // saturate diffusion coefficent
                if (c[k] < 0) {c[k] = 0;}
                else if (c[k] > 1) {c[k] = 1;}
		}
	}
         for (int i = 0; i < rows; i++) {
            for (int j = 0; j < cols; j++) {

                // current index
                k = i * cols + j;

                // diffusion coefficent
					cN = c[k];
					cS = c[iS[i] * cols + j];
					cW = c[k];
					cE = c[i * cols + jE[j]];

                // divergence (equ 58)
                D = cN * dN[k] + cS * dS[k] + cW * dW[k] + cE * dE[k];

                // image update (equ 61)
                J[k] = J[k] + 0.25*lambda*D;
            }
	}

#endif // CPU


#ifdef GPU

	//Currently the input size must be divided by 16 - the block size
	int block_x = cols/BLOCK_SIZE ;
    int block_y = rows/BLOCK_SIZE ;

    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(block_x , block_y);


	//Copy data from main memory to device memory
	//hipMemcpy(J_cuda, J, sizeof(float) * size_I, hipMemcpyHostToDevice);

	//Run kernels
	srad_cuda_1<<<dimGrid, dimBlock>>>(dE, dW, dN, dS, J, c, cols, rows, q0sqr);
  srad_cuda_2<<<dimGrid, dimBlock>>>(dE, dW, dN, dS, J, c, cols, rows, lambda, q0sqr);
  hipDeviceSynchronize();

	//Copy data from device memory to main memory
    //hipMemcpy(J, J_cuda, sizeof(float) * size_I, hipMemcpyDeviceToHost);

#endif
}

#ifdef GEM5_FUSION
    m5_work_end(0, 0);
#endif

    hipDeviceSynchronize();

#ifdef OUTPUT
    //Printing output
		printf("Printing Output:\n");
    for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
         printf("%.5f ", J[i * cols + j]);
		}
     printf("\n");
   }
#endif

	printf("Computation Done\n");

	free(I);
  free(J);
  free(dN); free(dS); free(dW); free(dE);
#ifdef CPU
	free(iN); free(iS); free(jW); free(jE);
#endif
#ifdef GPU
  //  hipFree(C_cuda);
	//hipFree(J_cuda);
	//hipFree(E_C);
	//hipFree(W_C);
	//hipFree(N_C);
	//hipFree(S_C);
#endif
	free(c);

}


void random_matrix(float *I, int rows, int cols){

	srand(7);

	for( int i = 0 ; i < rows ; i++){
		for ( int j = 0 ; j < cols ; j++){
		 I[i * cols + j] = rand()/(float)RAND_MAX ;
		}
	}

}

